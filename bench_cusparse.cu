#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSpMM
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

int generate_random_dense_matrix(int M, int N, float **outA, float density)
{
    int i, j;
    double rMax = (double)RAND_MAX;
    float *A = (float *)malloc(sizeof(float) * M * N);
    int totalNnz = 0;

    for (j = 0; j < N; j++)
    {
        for (i = 0; i < M; i++)
        {
            int r = rand();
            double dr = (double)r;
            float *curr = A + (j * M + i);

            if (dr / rMax > density)
            {
                *curr = 0.0f;
            }
            else
            {
                *curr = (dr / rMax) * 100.0;
            }

            if (*curr != 0.0f)
            {
                totalNnz++;
            }
        }
    }

    *outA = A;
    return totalNnz;
}

int main(void) {
    // Q, V: [seq-len, head-size] = [512, 64]
    // K: [head-size, seq-len] = [64, 512]
    // S: [seq-len, seq-len] = [512, 512]

    int SEQ_LEN = 512;
    int HEAD_SIZE = 64;
    float ATTN_DENSITY = 0.1;
    int QKV_SIZE = SEQ_LEN * HEAD_SIZE;

    int ldk = SEQ_LEN;
    int ldq = HEAD_SIZE;
    int ldv = SEQ_LEN;
    int ldo = SEQ_LEN;

    float alpha = 1.0f;
    float beta = 0.0f;

    float *dQ, *dV, *dK, *dS, *dCsrValS, *dO;
    float *hQ, *hV, *hK, *hS;
    int *dCsrRowPtrS, *dCsrColIndS, *dSNnzPerRow;
    void *dBuffer1, *dBuffer2, *dBuffer3;
    size_t bufferSize = 0;

    hipsparseSpMatDescr_t Sdescr;
    hipsparseDnMatDescr_t Kdescr, Qdescr, Vdescr, Odescr, SdescrDense;
    hipsparseHandle_t handle;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    float time_kernel;
    hipEvent_t start_event, stop_event;
    CHECK_CUDA(hipEventCreateWithFlags(&start_event, hipEventBlockingSync));
    CHECK_CUDA(hipEventCreateWithFlags(&stop_event, hipEventBlockingSync));

    size_t N_REPEAT = 10000;



    //// initialize dense Q, K, V, O
    hQ = (float *)calloc(QKV_SIZE, sizeof(float));
    hV = (float *)calloc(QKV_SIZE, sizeof(float));
    hK = (float *)calloc(QKV_SIZE, sizeof(float));

    CHECK_CUDA(hipMalloc((void **)&dQ, QKV_SIZE * sizeof(float)))
    CHECK_CUDA(hipMalloc((void**) &dV, QKV_SIZE * sizeof(float)))
    CHECK_CUDA(hipMalloc((void**) &dK, QKV_SIZE * sizeof(float)))
    CHECK_CUDA(hipMalloc((void**) &dO, QKV_SIZE * sizeof(float)))

    CHECK_CUDA(hipMemcpy(dQ, hQ, QKV_SIZE * sizeof(float), hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(dV, hV, QKV_SIZE * sizeof(float), hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(dK, hK, QKV_SIZE * sizeof(float), hipMemcpyHostToDevice))

    CHECK_CUSPARSE(hipsparseCreateDnMat(&Kdescr, SEQ_LEN, HEAD_SIZE, ldk, dK, HIP_R_32F, HIPSPARSE_ORDER_COL))
    CHECK_CUSPARSE(hipsparseCreateDnMat(&Qdescr, HEAD_SIZE, SEQ_LEN, ldq, dQ, HIP_R_32F, HIPSPARSE_ORDER_COL))
    CHECK_CUSPARSE(hipsparseCreateDnMat(&Vdescr, SEQ_LEN, HEAD_SIZE, ldv, dV, HIP_R_32F, HIPSPARSE_ORDER_COL))
    CHECK_CUSPARSE(hipsparseCreateDnMat(&Odescr, SEQ_LEN, HEAD_SIZE, ldo, dO, HIP_R_32F, HIPSPARSE_ORDER_COL))



    //// initialize sparsity mask S in CSR format
    int totalSNnz = generate_random_dense_matrix(SEQ_LEN, SEQ_LEN, &hS, ATTN_DENSITY);

    CHECK_CUDA(hipMalloc((void **)&dS, sizeof(float) * SEQ_LEN * SEQ_LEN));
    CHECK_CUDA(hipMalloc((void **)&dSNnzPerRow, sizeof(int) * SEQ_LEN));
    CHECK_CUDA(hipMalloc((void **)&dCsrValS, sizeof(float) * totalSNnz));
    CHECK_CUDA(hipMalloc((void **)&dCsrRowPtrS, sizeof(int) * (SEQ_LEN + 1)));
    CHECK_CUDA(hipMalloc((void **)&dCsrColIndS, sizeof(int) * totalSNnz));

    CHECK_CUDA(hipMemcpy(dS, hS, sizeof(float) * SEQ_LEN * SEQ_LEN, hipMemcpyHostToDevice));

    CHECK_CUSPARSE(hipsparseCreateDnMat(&SdescrDense, SEQ_LEN, SEQ_LEN, SEQ_LEN, dS, HIP_R_32F, HIPSPARSE_ORDER_ROW))
    CHECK_CUSPARSE(hipsparseCreateCsr(&Sdescr, SEQ_LEN, SEQ_LEN, 0,
                                     dCsrRowPtrS, NULL, NULL,
                                     HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F))
    
    CHECK_CUSPARSE(hipsparseDenseToSparse_bufferSize(handle, SdescrDense, Sdescr, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, &bufferSize))
    CHECK_CUDA(hipMalloc(&dBuffer3, bufferSize))
    CHECK_CUSPARSE(hipsparseDenseToSparse_analysis(handle, SdescrDense, Sdescr, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, dBuffer3))
    CHECK_CUSPARSE(hipsparseCsrSetPointers(Sdescr, dCsrRowPtrS, dCsrColIndS, dCsrValS))
    CHECK_CUSPARSE(hipsparseDenseToSparse_convert(handle, SdescrDense, Sdescr, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, dBuffer3))



    /// Allocate external buffers for SDDMM and SpMM
    CHECK_CUSPARSE(cusparseConstrainedGeMM_bufferSize(
        handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, Kdescr, Qdescr, &beta, Sdescr, 
        HIP_R_32F, &bufferSize))
    CHECK_CUDA(hipMalloc(&dBuffer1, bufferSize))

    // allocate an external buffer if needed
    CHECK_CUSPARSE(hipsparseSpMM_bufferSize(
        handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, Sdescr, Vdescr, &beta, Odescr, HIP_R_32F,
        HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize))
    CHECK_CUDA(hipMalloc(&dBuffer2, bufferSize))


    /// Run SDDMM and SpMM
    // warmup
    CHECK_CUSPARSE(cusparseConstrainedGeMM(handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, Kdescr, Qdescr, &beta, Sdescr,
        HIP_R_32F, dBuffer1))

    CHECK_CUSPARSE(hipsparseSpMM(handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, Sdescr, Vdescr, &beta, Odescr, HIP_R_32F,
        HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer2))


    CHECK_CUDA(hipEventRecord(start_event, 0));

    for (size_t i = 0; i < N_REPEAT; i++) {
        CHECK_CUSPARSE(cusparseConstrainedGeMM(handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, Kdescr, Qdescr, &beta, Sdescr,
            HIP_R_32F, dBuffer1))

        CHECK_CUSPARSE(hipsparseSpMM(handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, Sdescr, Vdescr, &beta, Odescr, HIP_R_32F,
            HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer2))
    }

    CHECK_CUDA(hipEventRecord(stop_event, 0));
    CHECK_CUDA(hipEventSynchronize(stop_event));
    CHECK_CUDA(hipEventElapsedTime(&time_kernel, start_event, stop_event));
    printf("kernel:\t\t%.4f ms\n", time_kernel / N_REPEAT);



    /// cleanup
    CHECK_CUSPARSE(hipsparseDestroySpMat(Sdescr))
    CHECK_CUSPARSE(hipsparseDestroyDnMat(Kdescr))
    CHECK_CUSPARSE(hipsparseDestroyDnMat(Qdescr))
    CHECK_CUSPARSE(hipsparseDestroyDnMat(Vdescr))
    CHECK_CUSPARSE(hipsparseDestroyDnMat(Odescr))
    CHECK_CUSPARSE(hipsparseDestroyDnMat(SdescrDense))
    CHECK_CUSPARSE(hipsparseDestroy(handle))

    CHECK_CUDA(hipFree(dQ))
    CHECK_CUDA(hipFree(dK))
    CHECK_CUDA(hipFree(dV))
    CHECK_CUDA(hipFree(dO))
    CHECK_CUDA(hipFree(dS))
    CHECK_CUDA(hipFree(dCsrColIndS))
    CHECK_CUDA(hipFree(dCsrRowPtrS))
    CHECK_CUDA(hipFree(dCsrValS))
    CHECK_CUDA(hipFree(dSNnzPerRow))
    CHECK_CUDA(hipFree(dBuffer1))
    CHECK_CUDA(hipFree(dBuffer2))



    return EXIT_SUCCESS;
}

